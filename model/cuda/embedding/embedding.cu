#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <vector>
#include <stdio.h>
#include <iostream>
#include <torch/torch.h>

#define BLOCK_SIZE 32
#define THREAD_SIZE 1

__global__ void embedding_forward_kernel(
    const float* table,
    const int32_t* indices,
    float* output,
    const int32_t seq_length,
    const int32_t embed_dim) {

    // Calculating the global thread index    
    const int thread_idx = (blockIdx.x * BLOCK_SIZE + threadIdx.x);
   
    // Calculating the token's value this thread must fetch
    const int thread_token_idx = thread_idx * THREAD_SIZE / embed_dim;
   
    // Check sequence position is within bounds
    if (thread_token_idx >= seq_length) {
       return;
    }

    // Finally, copying the elements into the output tensor
    const int output_idx = thread_idx * THREAD_SIZE;
    const int token_offset = output_idx % embed_dim;

    const int table_idx = indices[thread_token_idx] * embed_dim + token_offset;

    #pragma unroll
    for(int i = 0; i < THREAD_SIZE; i++){
        output[output_idx + i] = table[table_idx + i];
    }
}

__global__ void embedding_backward_kernel(
    const float* grad_output,
    const int32_t* indices,
    float* grad_table,
    const int32_t seq_length,
    const int32_t embed_dim,
    const int32_t num_embeddings,
    const int32_t total_elements,
    const int32_t padding_idx) {

    // Calculating the global thread index    
    const int thread_idx = (blockIdx.x * BLOCK_SIZE + threadIdx.x);
   
    // Calculating the token's value this thread must fetch
    const int thread_token_idx = thread_idx * THREAD_SIZE / embed_dim;
   
    // Checking that the sequence position is within bounds
    if (thread_token_idx >= seq_length) {
       return;
    }

    // Skipping the gradient update if this is the padding token!
    if (indices[thread_token_idx] == padding_idx) {
        return;
    }

    // Finally, copying the elements into the output tensor
    const int output_idx = thread_idx * THREAD_SIZE;
    const int token_offset = output_idx % embed_dim;

    const int table_idx = indices[thread_token_idx] * embed_dim + token_offset;

    #pragma unroll
    for(int i = 0; i < THREAD_SIZE; i++){
        atomicAdd(&grad_table[table_idx + i], grad_output[output_idx + i]);
    }
}

torch::Tensor embedding_forward_cuda(
    const torch::Tensor& indices,
    const torch::Tensor& table) {
    
    const int batch_size = indices.size(0);
    const int seq_length = indices.size(1);

    const int embed_dim = table.size(1);

    torch::Tensor output = torch::zeros({batch_size, seq_length, embed_dim}, table.device());

    // Creating CUDA streams
    std::vector<hipStream_t> streams(batch_size);

    // Creating PyTorch streams
    for(int b = 0; b < batch_size; b++){
        hipStreamCreate(&streams[b]);
    }

    // Configuring a 1D block
    const int total_elements = seq_length * embed_dim;

    // Calculating total blocks in the grid for 1024 threads / block
    const int num_blocks = total_elements / BLOCK_SIZE * THREAD_SIZE;

    for(int b = 0; b < batch_size; b++){
    
        // Call the CUDA kernel
        embedding_forward_kernel<<<num_blocks, BLOCK_SIZE, 0, streams[b]>>>(
            table.data_ptr<float>(),
            indices[b].data_ptr<int32_t>(),
            output[b].data_ptr<float>(),
            seq_length,
            embed_dim);

        hipStreamSynchronize(streams[b]);
    }

    // Handling the created streams
    for(int b = 0; b < batch_size; b++){

        hipStreamSynchronize(streams[b]);
        hipStreamDestroy(streams[b]);
    }

    return output;
}

torch::Tensor embedding_backward_cuda(
    const torch::Tensor& grad_output,
    const torch::Tensor& indices,
    const torch::Tensor& table,
    const int32_t padding_idx) {

    auto batch_size = indices.size(0);
    auto seq_length = indices.size(1);

    auto embed_dim = table.size(1);
    auto num_embeddings = table.size(0);

    // Creating CUDA streams
    std::vector<hipStream_t> streams(batch_size);

    // Creating PyTorch streams
    for(int b = 0; b < batch_size; b++){
        hipStreamCreate(&streams[b]);
    }

    auto grad_weight = torch::zeros({batch_size, num_embeddings, embed_dim}, grad_output.options());

    const int total_elements = seq_length * embed_dim;

    // Number of blocks needed to process all elements
    const int num_blocks = total_elements / (BLOCK_SIZE * THREAD_SIZE);

    for(int i = 0; i < batch_size; i++){

        embedding_backward_kernel<<<num_blocks, BLOCK_SIZE, 0, streams[i]>>>(
            grad_output[i].data_ptr<float>(),
            indices[i].data_ptr<int32_t>(),
            grad_weight[i].data_ptr<float>(),
            seq_length,
            embed_dim,
            num_embeddings,
            total_elements,
            padding_idx
        );
        
        hipStreamSynchronize(streams[i]);
    }

    for(int b = 0; b < batch_size; b++){

        hipStreamSynchronize(streams[b]);
        hipStreamDestroy(streams[b]);
    }

    return grad_weight.sum(0);
}